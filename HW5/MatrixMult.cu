#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

__global__ void gpu_mult_kernel(int* A, int* B, int* C, const int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int k = 0; k < n; k++)
	{
		C[row * n + col] += A[row * n + k] * B[k * n + col];
	}
}

void cpu_mult(int n, int* A, int* B, int* C);
int* allocate_matrix(int n);
void randomize_matrix(int n, int* A);
void print_matrix(int n, int* A);

int main(int argc, char * argv[])
{
	if (argc != 2)
	{
		printf("Error: Missing argument");
		return 0;
	}

	srand(time(NULL));

	int n = atoi(argv[1]);
	int m = n * n * sizeof(int);
	if (n <= 2)
	{
		printf("Error: Invalid Matrix Size");
		return 0;
	}

	int *X, *Y, *Zcpu, *Zgpu, *Zsgpu;
	int *X_d, *Y_d, *Zgpu_d, *Zsgpu_d;
	allocate_matrix(n);
	X = allocate_matrix(n);
	Y = allocate_matrix(n);
	Zcpu = allocate_matrix(n);
	Zgpu = allocate_matrix(n);
	Zsgpu = allocate_matrix(n);
	randomize_matrix(n, X);
	randomize_matrix(n, Y);

	// print X
	print_matrix(n, X);
	print_matrix(n, Y);
	cpu_mult(n, X, Y, Zcpu);
	print_matrix(n, Zcpu);

	// allocate memory on gpu
	hipMalloc((void **)&X_d, m);
	hipMalloc((void **)&Y_d, m);
	hipMalloc((void **)&Zgpu_d, m);

	// copy host data to gpu
	hipMemcpy(X_d, X, m, hipMemcpyHostToDevice);
	hipMemcpy(Y_d, Y, m, hipMemcpyHostToDevice);
	hipMemcpy(Zgpu_d, Zcpu, m, hipMemcpyHostToDevice);

	// kernel parameters
	dim3 dimGrid(n / 2, n / 2, 1);
	dim3 dimBlock(2, 2, 1);

	// run kernel
	gpu_mult_kernel << <dimGrid, dimBlock >> > (X_d, Y_d, Zgpu_d, n);

	// copy result back
	hipMemcpy(Zgpu, Zgpu_d, m, hipMemcpyDeviceToHost);
	print_matrix(n, Zgpu);
}

void cpu_mult(int n, int* A, int* B, int* C)
{
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			for (int k = 0; k < n; k++)
				C[i * n + j] += A[i * n + k] * B[k * n + j];
}

int* allocate_matrix(int n)
{
	int* A = (int *)malloc(n * n * sizeof(int*));
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			A[i * n + j] = 0;
	return A;
}

void randomize_matrix(int n, int * A)
{
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			A[i * n + j] = rand() % 10;
}

void print_matrix(int n, int* A)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
			printf("%4d ", A[i * n + j]);
		printf("\n");
	}
	printf("\n");
}










